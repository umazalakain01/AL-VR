#include "hip/hip_runtime.h"
#include <stdint.h>

#include "RGBToNV12.h"

__device__ float rgb2y(uchar4 c) {
	return 0.257f * c.x + 0.504f * c.y + 0.098f * c.z + 16.0f;
}

__device__ float rgb2u(uchar4 c) {
	return -0.148f * c.x - 0.291f * c.y + 0.439f * c.z + 128.0f;
}

__device__ float rgb2v(uchar4 c) {
	return 0.439f * c.x - 0.368f * c.y - 0.071f * c.z + 128.0f;
}

texture<uchar4, hipTextureType2D, hipReadModeElementType> texRef;

__global__ void RGBA2NV12_kernel(uint8_t *dstImage, size_t destPitch,
	uint32_t width, uint32_t height)
{
	// Pad borders with duplicate pixels, and we multiply by 2 because we process 2 pixels per thread
	int32_t x = blockIdx.x * (blockDim.x << 1) + (threadIdx.x << 1);
	int32_t y = blockIdx.y * (blockDim.y << 1) + (threadIdx.y << 1);

	int x1 = x + 1;
	int y1 = y + 1;

	if (x1 >= width)
		return; //x = width - 1;

	if (y1 >= height)
		return; // y = height - 1;

	uchar4 c00 = tex2D(texRef, x, y);
	uchar4 c01 = tex2D(texRef, x1, y);
	uchar4 c10 = tex2D(texRef, x, y1);
	uchar4 c11 = tex2D(texRef, x1, y1);

	uint8_t y00 = (uint8_t)(rgb2y(c00) + 0.5f);
	uint8_t y01 = (uint8_t)(rgb2y(c01) + 0.5f);
	uint8_t y10 = (uint8_t)(rgb2y(c10) + 0.5f);
	uint8_t y11 = (uint8_t)(rgb2y(c11) + 0.5f);

	uint8_t u = (uint8_t)((rgb2u(c00) + rgb2u(c01) + rgb2u(c10) + rgb2u(c11)) * 0.25f + 0.5f);
	uint8_t v = (uint8_t)((rgb2v(c00) + rgb2v(c01) + rgb2v(c10) + rgb2v(c11)) * 0.25f + 0.5f);

	dstImage[destPitch * y + x] = y00;
	dstImage[destPitch * y + x1] = y01;
	dstImage[destPitch * y1 + x] = y10;
	dstImage[destPitch * y1 + x1] = y11;

	uint32_t chromaOffset = destPitch * height;
	int32_t x_chroma = x;
	int32_t y_chroma = y >> 1;

	dstImage[chromaOffset + destPitch * y_chroma + x_chroma] = u;
	dstImage[chromaOffset + destPitch * y_chroma + x_chroma + 1] = v;
}

extern "C"
hipError_t RGBA2NV12(hipArray *srcImage,
	uint8_t *dstImage, size_t destPitch,
	uint32_t width, uint32_t height)
{
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(8, 8, 8, 8, hipChannelFormatKindUnsigned);

	// Set texture parameters
	texRef.addressMode[0] = hipAddressModeWrap;
	texRef.addressMode[1] = hipAddressModeWrap;
	texRef.filterMode = hipFilterModePoint;
	texRef.normalized = false;

	hipError_t cudaStatus = hipBindTextureToArray(texRef, srcImage, channelDesc);
	if (cudaStatus != hipSuccess) {
		return cudaStatus;
	}

	dim3 block(32, 16, 1);
	dim3 grid((width + (2 * block.x - 1)) / (2 * block.x), (height + (2 * block.y - 1)) / (2 * block.y), 1);

	RGBA2NV12_kernel<<<grid, block>>>(dstImage, destPitch, width, height);

	hipDeviceSynchronize();

	cudaStatus = hipGetLastError();
	return cudaStatus;
}